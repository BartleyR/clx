#include "hip/hip_runtime.h"
#include <vector>
#include <string>

#include "cp_util.cuh"
#include "data_transfer_utils.cuh"
#include "hipcub/hipcub.hpp"
#include "tokenizer_utils.cuh"
#include "tokenizers.cuh"

#define SORT_BIT 22
#define THREADS_PER_BLOCK 64

/*
  Returns true if the byte passed in could be a valid head byte for
  a utf8 character.
*/
__device__ __forceinline__ bool is_head_byte(unsigned char utf8_byte){
  return (utf8_byte >> 6) != 2;
}


/*
  If the byte at start_byte_for_thread is a head byte, the unicode code-point encoded by
  the utf8 character started at that byte is returned and the head_byte boolean passed in
  is set to true.

  If the byte at start_byte_for_thread is not a head byte, 0 is returned AND the head_byte
  boolean passed in is set to false.

  All threads start reading bytes from the pointer denoted by sentences.

  Params
  --------
  sentences: A pointer to the start of the sequence of characters to be tokenized.
*/
__device__ __forceinline__ uint32_t extract_code_points_from_utf8(const unsigned char* sentences,
                                                                  const uint32_t start_byte_for_thread,
                                                                  bool& head_byte) {

  constexpr uint8_t max_utf8_blocks_for_char = 4;
  uint8_t utf8_blocks[max_utf8_blocks_for_char];

  #pragma unroll 
  for(int i = 0; i < max_utf8_blocks_for_char; ++i) {
    utf8_blocks[i] = sentences[start_byte_for_thread + i];
  }
  
  // We can have at most 5 bits encoding the length. We check those bits to infer the actual length
  const uint8_t length_encoding_bits = utf8_blocks[0] >> 3;

  head_byte = is_head_byte(utf8_blocks[0]);

  // Set the number of characters and the top masks based on the
  // length encoding bits.
  uint8_t char_encoding_length = 0, top_mask = 0;
  if (length_encoding_bits < 16){
    char_encoding_length = 1;
    top_mask = 0x7F;
  } else if (length_encoding_bits >= 24 && length_encoding_bits <= 27) {
    char_encoding_length = 2;
    top_mask = 0x1F;
  } else if (length_encoding_bits == 28 || length_encoding_bits == 29) {
    char_encoding_length = 3;
    top_mask = 0x0F;
  } else if (length_encoding_bits == 30) {
    char_encoding_length = 4;
    top_mask = 0x07;
  }

  // Now pack up the bits into a uint32_t. All threads will process 4 bytes
  // to reduce divergence.
  uint32_t code_point = (utf8_blocks[0] & top_mask) << 18;

  #pragma unroll
  for(int i = 1; i < 4; ++i) {
    code_point |= ((utf8_blocks[i] & 0x3F) << (18 - 6*i));
  }

  // Zero out the bottom of code points with extra reads
  const uint8_t shift_amt = 24 - 6*char_encoding_length;
  code_point >>= shift_amt;

  return code_point;
}

__global__ void gpuBasicTokenizer(const unsigned char* sentences,  uint32_t* device_sentence_offsets,
                                  const size_t total_bytes, uint32_t* cp_metadata, uint64_t* aux_table, 
                                  uint32_t* code_points, uint32_t* chars_per_thread, bool do_lower_case,
                                  uint32_t num_sentences) {

  constexpr uint32_t init_val = (1 << SORT_BIT);
  uint32_t replacement_code_points[MAX_NEW_CHARS] = {init_val, init_val, init_val};
  
  bool head_byte = false;
  const uint32_t char_for_thread = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t num_new_chars = 0;                                  

  if(char_for_thread < total_bytes){                                  
    const uint32_t code_point = extract_code_points_from_utf8(sentences, char_for_thread, head_byte);
    const uint32_t thr_cp_metadata = get_cp_metadata(cp_metadata, code_point);

    if(!should_remove_cp(thr_cp_metadata, do_lower_case) && head_byte) {
      num_new_chars = 1;
      // Apply lower cases and accent stripping if necessary
      const bool replacement_needed = do_lower_case || always_replace(thr_cp_metadata);
      uint32_t new_cp = replacement_needed? get_first_cp(thr_cp_metadata): code_point;
      new_cp = new_cp == 0? code_point: new_cp;

      replacement_code_points[0] = new_cp;
      if(is_multi_char_transform(thr_cp_metadata) && do_lower_case) {
        uint64_t next_cps = get_extra_cps(aux_table, code_point);
        replacement_code_points[1] = static_cast<uint32_t>(next_cps >> 32);
        const uint32_t potential_next_cp = static_cast<uint32_t>(next_cps);
        replacement_code_points[2] = potential_next_cp != 0? potential_next_cp: replacement_code_points[2];
        num_new_chars = 2 + (potential_next_cp != 0);
      }

      if(should_add_spaces(thr_cp_metadata, do_lower_case)){

        // Need to shift all existing code-points up one
        for(int loc = num_new_chars; loc > 0; --loc) {
          replacement_code_points[loc] = replacement_code_points[loc - 1];
        }

        // Write the required spaces at the end
        replacement_code_points[0] = SPACE_CODE_POINT;
        replacement_code_points[num_new_chars + 1] = SPACE_CODE_POINT;
        num_new_chars += 2;
      }
    }
  }

  chars_per_thread[char_for_thread] = num_new_chars;
    
  typedef hipcub::BlockStore<uint32_t, THREADS_PER_BLOCK, MAX_NEW_CHARS, hipcub::BLOCK_STORE_WARP_TRANSPOSE> BlockStore;
  __shared__ typename BlockStore::TempStorage temp_storage;

  // Now we perform coalesced writes back to global memory using cub.
  uint32_t* block_base = code_points + blockIdx.x * blockDim.x * MAX_NEW_CHARS;
  BlockStore(temp_storage).Store(block_base, replacement_code_points);
}


void transfer_cp_data_to_device(uint32_t **device_cp_metadata, uint64_t **device_aux_data) {
  malloc_and_copy_vec_to_device(device_cp_metadata, cp_data);
  malloc_and_copy_vec_to_device(device_aux_data, aux_data);
}


void flatten_sentences(const std::vector<std::string>& sentences,
                       char* flattened_sentences, 
                       uint32_t* sentence_offsets) {

  uint32_t start_copy = 0;
  for(int i = 0; i < sentences.size(); ++i){
    const uint32_t sentence_length = sentences[i].size();

    sentences[i].copy(flattened_sentences + start_copy, sentence_length);
    sentence_offsets[i] = start_copy;
    start_copy += sentence_length;
  }
  sentence_offsets[sentences.size()] = start_copy;
}


// -------------------------------------- Basic tokenizer definitions ------------------------------------------------------------
// See tokenizers.cuh

GpuBasicTokenizer::GpuBasicTokenizer(uint32_t max_num_sentences, uint32_t max_num_chars, bool do_lower_case):
  do_lower_case(do_lower_case), device_cp_metadata(nullptr), device_aux_table(nullptr) {
  transfer_cp_data_to_device(&device_cp_metadata, &device_aux_table);
  assertCudaSuccess(hipMalloc(&device_sentence_offsets, sizeof(*device_sentence_offsets) * max_num_sentences + 1));
  assertCudaSuccess(hipMalloc(&device_sentences, sizeof(*device_sentences) * max_num_chars));

  size_t max_BLOCKS = (max_num_chars + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  size_t max_threads_on_device = max_BLOCKS * THREADS_PER_BLOCK;

  const size_t max_new_char_total = MAX_NEW_CHARS * max_threads_on_device;
  const size_t device_code_points_size = sizeof(*device_code_points) * max_new_char_total;
  assertCudaSuccess(hipMalloc(&device_code_points, device_code_points_size));

  const size_t device_chars_per_thread_size = sizeof(*device_chars_per_thread) * max_threads_on_device;
  assertCudaSuccess(hipMalloc(&device_chars_per_thread, device_chars_per_thread_size));

  // Determine temporary device storage requirements for cub
  size_t temp_storage_scan_bytes = 0;
  uint32_t* device_chars_per_thread = nullptr;
  hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_scan_bytes, device_chars_per_thread, device_chars_per_thread, max_threads_on_device);
  size_t temp_storage_select_bytes = 0;
  static NotEqual select_op((1 << SORT_BIT));
  hipcub::DeviceSelect::If(nullptr, temp_storage_select_bytes, device_code_points, device_code_points, 
                        device_num_selected, max_new_char_total, select_op);
  max_cub_storage_bytes = std::max(temp_storage_scan_bytes, temp_storage_select_bytes);
  assertCudaSuccess(hipMalloc(&cub_temp_storage, max_cub_storage_bytes));
  const size_t device_num_selected_size = sizeof(*device_num_selected);
  assertCudaSuccess(hipMalloc(&device_num_selected, device_num_selected_size));
}



std::pair<ptr_length_pair<uint32_t*>, ptr_length_pair<uint32_t*>> GpuBasicTokenizer::tokenize(const std::vector<std::string>& sentences) {

  ptr_length_pair<uint32_t*> cp_and_length;
  ptr_length_pair<uint32_t*> offset_and_length;

  size_t total_sentence_bytes = 0;                            
  for(const auto& sentence: sentences) {
    total_sentence_bytes += sentence.length();
  }

  size_t num_offsets = sentences.size() + 1;
  uint32_t* sentence_offsets = new uint32_t[num_offsets];
  char* flattened_sentences = new char[total_sentence_bytes];
  flatten_sentences(sentences, flattened_sentences, sentence_offsets);
  assertCudaSuccess(hipMemcpy(device_sentence_offsets, sentence_offsets, sizeof(*device_sentence_offsets) * num_offsets, hipMemcpyHostToDevice));
  assertCudaSuccess(hipMemcpy(device_sentences, flattened_sentences, total_sentence_bytes, hipMemcpyHostToDevice));

  static NotEqual select_op((1 << SORT_BIT));
  
  size_t BLOCKS = (total_sentence_bytes + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  const size_t max_new_char_total = MAX_NEW_CHARS * BLOCKS * THREADS_PER_BLOCK;
  size_t threads_on_device = BLOCKS * THREADS_PER_BLOCK;

  gpuBasicTokenizer<<<BLOCKS, THREADS_PER_BLOCK>>>(device_sentences, device_sentence_offsets, total_sentence_bytes, device_cp_metadata, device_aux_table, 
                                                   device_code_points, device_chars_per_thread, do_lower_case, sentences.size()); 
  assertCudaSuccess(hipPeekAtLastError());                                    

  hipcub::DeviceSelect::If(cub_temp_storage, max_cub_storage_bytes, device_code_points, device_code_points, device_num_selected, max_new_char_total, select_op);
  assertCudaSuccess(hipPeekAtLastError());

  // We also need to prefix sum the number of characters up to an including the current character in order to get the new sentence lengths.
  hipcub::DeviceScan::InclusiveSum(cub_temp_storage, max_cub_storage_bytes, device_chars_per_thread, device_chars_per_thread, threads_on_device);  
  assertCudaSuccess(hipPeekAtLastError());

  constexpr uint16_t SENTENCE_UPDATE_THREADS = 64;                              
  size_t SEN_KERNEL_BLOCKS = (sentences.size() + SENTENCE_UPDATE_THREADS - 1) / SENTENCE_UPDATE_THREADS;   
  update_sentence_lengths<<<SEN_KERNEL_BLOCKS, SENTENCE_UPDATE_THREADS>>>(device_sentence_offsets, device_chars_per_thread, sentences.size());
  assertCudaSuccess(hipPeekAtLastError());   

  offset_and_length.gpu_ptr = device_sentence_offsets;
  offset_and_length.length = sentences.size() + 1;

  uint32_t num_chars = 0;
  assertCudaSuccess(hipMemcpy(&num_chars, offset_and_length.gpu_ptr + sentences.size(), sizeof(num_chars), hipMemcpyDeviceToHost));
  cp_and_length.gpu_ptr = device_code_points;
  cp_and_length.length = num_chars;
  
  return std::make_pair(cp_and_length, offset_and_length);
}

std::pair<ptr_length_pair<uint32_t*>, ptr_length_pair<uint32_t*>> GpuBasicTokenizer::tokenize(const char* device_sentences_, uint32_t* offsets, uint32_t offset_size)  {

  ptr_length_pair<uint32_t*> cp_and_length;
  ptr_length_pair<uint32_t*> offset_and_length;

  size_t num_offsets = offset_size + 1;
  uint32_t* sentence_offsets = new uint32_t[num_offsets];
  uint32_t start_copy = 0;
  for(int i = 0; i < offset_size; ++i){
    sentence_offsets[i] = start_copy;
    start_copy += offsets[i];
  }
  sentence_offsets[offset_size] = start_copy;

  assertCudaSuccess(hipMemcpy(device_sentence_offsets, sentence_offsets, sizeof(*device_sentence_offsets) * num_offsets, hipMemcpyHostToDevice));

  static NotEqual select_op((1 << SORT_BIT));

  size_t BLOCKS = (sentence_offsets[offset_size] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  const size_t max_new_char_total = MAX_NEW_CHARS * BLOCKS * THREADS_PER_BLOCK;
  size_t threads_on_device = BLOCKS * THREADS_PER_BLOCK;

  gpuBasicTokenizer<<<BLOCKS, THREADS_PER_BLOCK>>>((unsigned char*)device_sentences_, device_sentence_offsets, sentence_offsets[offset_size], device_cp_metadata, device_aux_table,
                                                   device_code_points, device_chars_per_thread, do_lower_case, offset_size);
  assertCudaSuccess(hipPeekAtLastError());

  hipcub::DeviceSelect::If(cub_temp_storage, max_cub_storage_bytes, device_code_points, device_code_points, device_num_selected, max_new_char_total, select_op);
  assertCudaSuccess(hipPeekAtLastError());

  // We also need to prefix sum the number of characters up to an including the current character in order to get the new sentence lengths.
  hipcub::DeviceScan::InclusiveSum(cub_temp_storage, max_cub_storage_bytes, device_chars_per_thread, device_chars_per_thread, threads_on_device);
  assertCudaSuccess(hipPeekAtLastError());

  constexpr uint16_t SENTENCE_UPDATE_THREADS = 64;
  size_t SEN_KERNEL_BLOCKS = (offset_size + SENTENCE_UPDATE_THREADS - 1) / SENTENCE_UPDATE_THREADS;
  update_sentence_lengths<<<SEN_KERNEL_BLOCKS, SENTENCE_UPDATE_THREADS>>>(device_sentence_offsets, device_chars_per_thread, offset_size);
  assertCudaSuccess(hipPeekAtLastError());

  offset_and_length.gpu_ptr = device_sentence_offsets;
  offset_and_length.length = offset_size + 1;

  uint32_t num_chars = 0;
  assertCudaSuccess(hipMemcpy(&num_chars, offset_and_length.gpu_ptr + offset_size, sizeof(num_chars), hipMemcpyDeviceToHost));
  cp_and_length.gpu_ptr = device_code_points;
  cp_and_length.length = num_chars;

  return std::make_pair(cp_and_length, offset_and_length);
}

GpuBasicTokenizer::~GpuBasicTokenizer() {
  assertCudaSuccess(hipFree(device_aux_table));
  assertCudaSuccess(hipFree(device_cp_metadata));
  assertCudaSuccess(hipFree(device_sentences));
  assertCudaSuccess(hipFree(device_sentence_offsets));
  assertCudaSuccess(hipFree(device_code_points));
  assertCudaSuccess(hipFree(device_chars_per_thread));
  assertCudaSuccess(hipFree(device_num_selected));
  assertCudaSuccess(hipFree(cub_temp_storage));
}
